
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include <__clang_cuda_math.h>
#include <cstdlib>
#include <iostream>
#include <stdexcept>

__global__ void add(int n, float *a, float *b, float *c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

__global__ void forward(int batch_size, int n, int out_w, float *input,
                        float *weights, float *biases, float *output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < batch_size && col < out_w) {
    output[row * out_w + col] = biases[col];
    for (int i = 0; i < n; i++) {
      output[row * out_w + col] +=
          weights[i * out_w + col] * input[row * n + i];
    }
  }
}

__global__ void relu(int w, int h, float *input, float *output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < h && col < w) {
    float activation = input[row * w + col];
    output[row * w + col] = activation > 0.f ? activation : 0.f;
  }
}

__global__ void softmax(int w, int h, float *input, float *output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < h && col < w) {
    float maxval = input[row * w];
    for (int i = 1; i < w; i++) {
      maxval = max(maxval, input[row * w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i < w; i++) {
      divisor += exp(input[row * w + i] - maxval);
    }
    output[row * w + col] = exp(input[row * w + col] - maxval) / divisor;
  }
}

__global__ void cross_entropy(int w, int h, float *preds, float *real,
                              float *output) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < h) {
    float loss = 0.f;
    for (int i = 0; i < w; i++) {
      loss -= real[idx * w + i] * log(max(1e-6, preds[idx * w + i]));
    }
    output[idx] = loss;
  }
}

__global__ void init_rand(int w, int h, float *mat) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < h && col < w) {
    hiprandState state;
    hiprand_init(42, row * w + col, 0, &state);
    mat[row * w + col] = hiprand_uniform(&state) * sqrtf(2.f / h);
  }
}

#define cudaSafeCall(call)                                                     \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << ": "     \
                << hipGetErrorString(error) << std::endl;                     \
      throw std::runtime_error("CUDA Error");                                  \
    }                                                                          \
  } while (0)

int main(void) {
  try {
    int N = 4096;

    float *a = (float *)malloc(N * sizeof(float));
    float *b = (float *)malloc(N * sizeof(float));
    float *c = (float *)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
      a[i] = (float)rand() / (float)RAND_MAX;
      b[i] = (float)rand() / (float)RAND_MAX;
    }

    float *a_d;
    float *b_d;
    float *c_d;
    cudaSafeCall(hipMalloc((void **)&a_d, N * sizeof(float)));
    cudaSafeCall(hipMalloc((void **)&b_d, N * sizeof(float)));
    cudaSafeCall(hipMalloc((void **)&c_d, N * sizeof(float)));

    cudaSafeCall(hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(b_d, b, N * sizeof(float), hipMemcpyHostToDevice));

    int BLOCK_SIZE = 1024;
    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    add<<<gridSize, BLOCK_SIZE>>>(N, a_d, b_d, c_d);
    cudaSafeCall(hipPeekAtLastError());
    cudaSafeCall(hipDeviceSynchronize());

    cudaSafeCall(hipMemcpy(c, c_d, N * sizeof(float), hipMemcpyDeviceToHost));

    cudaSafeCall(hipFree(a_d));
    cudaSafeCall(hipFree(b_d));
    cudaSafeCall(hipFree(c_d));
    free(a);
    free(b);
    free(c);
    hipDeviceReset();
    return EXIT_SUCCESS;
  } catch (const std::exception &e) {
    std::cerr << "Error: " << e.what() << std::endl;
    return EXIT_FAILURE;
  }
}
